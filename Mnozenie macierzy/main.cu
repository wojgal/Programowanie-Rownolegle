#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N 1024
#define BLOCK_SIZE 16

struct BlockStatus {
    int state;
};

__global__ void matrixMultiplication(float *A, float *B, float *C, BlockStatus *status)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int blockRow = row / BLOCK_SIZE;
    int blockCol = col / BLOCK_SIZE;
    int blockIndex = blockRow * (N / BLOCK_SIZE) + blockCol;

    while (status[blockIndex].state != 2)
    {
        // Oczekiwanie na zakończenie pobierania danych przez inny blok
        __threadfence();
    }

    float sum = 0.0f;
    for (int i = 0; i < N; i++)
    {
        sum += A[row * N + i] * B[i * N + col];
    }

    C[row * N + col] = sum;

    if (threadIdx.x == 0 && threadIdx.y == 0)
    {
        // Aktualizacja stanu bloku wątków
        atomicExch(&status[blockIndex].state, 2);
    }

    // Synchronizacja wszystkich wątków w bloku przed kontynuacją
    __threadfence();
}

void initializeMatrices(float* A, float* B, int size)
{
    srand(time(NULL));

    for (int i = 0; i < size; i++)
    {
        A[i] = static_cast<float>(rand()) / RAND_MAX;
        B[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

int main()
{
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;
    int size = N * N * sizeof(float);

    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    initializeMatrices(A, B, N * N);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    BlockStatus *d_status;
    hipMalloc(&d_status, sizeof(BlockStatus) * (N / BLOCK_SIZE) * (N / BLOCK_SIZE));
    hipMemset(d_status, 0, sizeof(BlockStatus) * (N / BLOCK_SIZE) * (N / BLOCK_SIZE));

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize(N / blockSize.x, N / blockSize.y);

    matrixMultiplication<<<gridSize, blockSize>>>(d_A, d_B, d_C, d_status);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_status);

    return 0;
}
